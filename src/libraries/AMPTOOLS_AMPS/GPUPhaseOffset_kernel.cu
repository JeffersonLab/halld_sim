#include "hip/hip_runtime.h"

#include <stdio.h>

#include "GPUManager/GPUCustomTypes.h"
#include "GPUManager/CUDA-Complex.cuh"


__global__ void
GPUPhaseOffset_kernel(GPU_AMP_PROTO, GDouble phase)
{

  WCUComplex ans = {cos(phase), sin(phase)};  
  pcDevAmp[GPU_THIS_EVENT] = ans;

}

void
GPUPhaseOffset_exec(dim3 dimGrid, dim3 dimBlock, GPU_AMP_PROTO, GDouble m_phase)
{
  GPUPhaseOffset_kernel<<< dimGrid, dimBlock >>>(GPU_AMP_ARGS, m_phase);
}
