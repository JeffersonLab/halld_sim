#include "hip/hip_runtime.h"

#include <stdio.h>

#include "GPUManager/GPUCustomTypes.h"
#include "GPUManager/CUDA-Complex.cuh"

#include "GPUUtils/wignerD.cuh"
#include "GPUUtils/clebsch.cuh"
 
__global__ void
GPUTwoPSMoment_kernel(GPU_AMP_PROTO, GDouble *H, int *alpha, int *L, int *M, int nMoments )
{
    int iEvent = GPU_THIS_EVENT;

    GDouble pGamma = GPU_UVARS(0);
    GDouble cosTheta = GPU_UVARS(1);
    GDouble phi = GPU_UVARS(2);
    GDouble bigPhi = GPU_UVARS(3);

    GDouble total = 0;
    for(int imom = 0; imom < nMoments; imom++) { 
	   	   
	    int Galpha = alpha[imom];
	    int GL = L[imom];
	    int GM = M[imom];
	    
	    GDouble mom = 2.0 * sqrt( (2*GL + 1) / (4*PI ) );
	    if(Galpha == 0)
		   mom *= Y( GL, GM, cosTheta, phi ).m_dRe;
	    else if(Galpha == 1) 
		    mom *= pGamma * cos(2*bigPhi) * Y( GL, GM, cosTheta, phi ).m_dRe;
	    else if(Galpha == 2) 
		    mom *= -1 * pGamma * sin(2*bigPhi) * Y( GL, GM, cosTheta, phi ).m_dIm;
		    
	    // m = 0 only non-zero for alpha = 0, 1 but half the size of other m-projections
	    if(GM == 0 && Galpha < 2) mom *= 0.5;
		    
	    total += H[imom]*mom;
    }   

    WCUComplex amp = { sqrt( fabs( total ) ), 0 };

    pcDevAmp[iEvent] = amp;
}

void
GPUTwoPSMoment_exec(dim3 dimGrid, dim3 dimBlock, GPU_AMP_PROTO, GDouble* H, int* alpha, int* L, int *M, int nMoments)
{

  // allocate memory and pass moment parameter array to GPU
  GDouble* d_H;
  int *d_alpha, *d_L, *d_M;
  hipMalloc((void**)&d_H, nMoments * sizeof(GDouble));
  hipMalloc((void**)&d_alpha, nMoments * sizeof(int));
  hipMalloc((void**)&d_L, nMoments * sizeof(int));
  hipMalloc((void**)&d_M, nMoments * sizeof(int));
  hipMemcpy(d_H, &H[0], nMoments * sizeof(GDouble), hipMemcpyHostToDevice );
  hipMemcpy(d_alpha, &alpha[0], nMoments * sizeof(int), hipMemcpyHostToDevice );
  hipMemcpy(d_L, &L[0], nMoments * sizeof(int), hipMemcpyHostToDevice );
  hipMemcpy(d_M, &M[0], nMoments * sizeof(int), hipMemcpyHostToDevice );

  GPUTwoPSMoment_kernel<<< dimGrid, dimBlock >>>(GPU_AMP_ARGS, d_H, d_alpha, d_L, d_M, nMoments);

  hipDeviceSynchronize();
  hipFree(d_H);
}
