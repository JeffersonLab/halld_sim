#include "hip/hip_runtime.h"
/*
 *  GPUb1piAngAmp_kernel.cu
 *
 */

#include <stdio.h>
#include "hip/hip_runtime.h"

//  Original headers were scattered around file system
#include "GPUManager/GPUCustomTypes.h"
#include "GPUManager/CUDA-Complex.cuh"

#include "GPUUtils/lorentzBoost.cuh"
#include "GPUUtils/threeVector.cuh"
#include "GPUUtils/wignerD.cuh"
#include "GPUUtils/clebsch.cuh"

#include "AMPTOOLS_AMPS/breakupMomentum.cuh"
#include "AMPTOOLS_AMPS/barrierFactor.cuh"


//  Test headers
#if 0
#include "GPUCustomTypes.h"
#include "CUDA-Complex.cuh"

#include "lorentzBoost.cuh"
#include "threeVector.cuh"
#include "wignerD.cuh"
#include "clebsch.cuh"

#include "breakupMomentum.cuh"
#include "barrierFactor.cuh"
#endif



#define ADD4(a,b) { a[0]+b[0], a[1]+b[1], a[2]+b[2], a[3]+b[3] }

#define MASS(v)   (G_SQRT(v[0]*v[0]-v[1]*v[1]-v[2]*v[2]-v[3]*v[3]))

#define Nterm(J)  (G_SQRT((2*J+1)/(4*M_PI)))


// Macro to ease definition of loops
#define LOOP(INDEX,START,END,INC) for (int INDEX=START;INDEX<=END;INDEX+=INC)


static __device__ void //note: 4-vector input presumed
rotateZ( GDouble* v, GDouble phi ){
  GDouble sinphi = G_SIN(phi);
  GDouble cosphi = G_COS(phi);
  GDouble tx;
  tx   = v[1] * cosphi - v[2] * sinphi;
  v[2] = v[2] * cosphi + v[1] * sinphi;
  v[1] = tx;
}

static __device__ void //note: 4-vector input presumed
rotateY ( GDouble* v, GDouble theta) {
  double sinphi = G_SIN(theta);
  double cosphi = G_COS(theta);
  double tz;
  tz = v[3] * cosphi - v[1] * sinphi;
  v[1] = v[1] * cosphi + v[3] * sinphi;
  v[3] = tz;
}

static __device__ GDouble  //note: 3-vector input presumed
theta( GDouble* pv ){
  GDouble r= G_SQRT(pv[0]*pv[0] + pv[1]*pv[1]);
  return  G_ATAN2( r , pv[2] );
}


static __device__ void
MoveToRF(GDouble *parent, GDouble *daughter)
{
  GDouble *par3vec=parent+1;
  rotateZ( daughter , -phi(par3vec) );
  rotateY( daughter , -theta(par3vec) );

  GDouble beta[]={0,0, -G_SQRT(dot(par3vec,par3vec))/parent[0]};
  //** (x)  Might this be bootToRest???
  // beta is defined to boost to parent's rest frame
  // I just adapted GPUUtil boost fcn with vector beta input
  boost( daughter , beta );

}



static __device__ WCUComplex
BreitWigner_loc(GDouble m0, GDouble Gamma0, int L,
                GDouble *P1, GDouble *P2)
{
  
  GDouble  Ptot[4] = ADD4(P1, P2);  
  GDouble m  = MASS(Ptot);
  GDouble mass1 = MASS(P1);
  GDouble mass2 = MASS(P2);
  
  
  // assert positive breakup momenta     
  GDouble q0 = fabs( breakupMomentum(m0, mass1, mass2) );
  GDouble q  = fabs( breakupMomentum(m,  mass1, mass2) );
  
  //printf("BW: (%5.3f, %5.3f, %d) m=%6.4f m1=%6.4f m2=%6.4f q=%6.4f q0=%6.4f\n",
  //  m0,Gamma0,L,m,mass1,mass2,q,q0);
  
  GDouble F0 = L==0 ? 1.0 : barrierFactor(q0, L);
  GDouble F  = L==0 ? 1.0 : barrierFactor(q,  L);
  
  GDouble width_coef=Gamma0*(m0/m);
  //GDouble qq0=q/q0;
  //GDouble width_qdep = (L==0 ? qq0 : (L==1 ? qq0*qq0*qq0 : pow(qq0,2*L+1)))*((F*F)/(F0*F0));
  GDouble width_qdep = q/q0  * (F*F)/(F0*F0);
  //GDouble num_qdep = (L==0 ? q : (L==1 ? q*q*q : pow(q,2*L+1)))*(F*F);
  GDouble num_qdep = q*(F*F);
  
  GDouble width = width_coef * width_qdep;
  
  //complex<GDouble> bwtop(m0 * width, 0.0 );
  WCUComplex bwtop = { G_SQRT(m0*width_coef) * num_qdep, 0 };
  
  WCUComplex bwbottom =  { m0*m0 - m*m  , -1.0 * ( m0 * width ) };
  
  return  ( bwtop / bwbottom );
  
}


//  JR 2012-07-29
//  Set all Amplitudes to 0 on the Device.  This is needed now because we only
//  calculate amplitudes for those momenta sets with non-zero amplitudes.  If
//  this function were not performed, amplitudes which are supposed to be zero will
//  be undefined.
__global__ void Setzero_kernel(WCUComplex *pcDevAmp, int iNEvents) {
  int iEvent = GPU_THIS_EVENT;
  if (iEvent>=iNEvents) return;
  pcDevAmp[iEvent].m_dRe = 0.0;
  pcDevAmp[iEvent].m_dIm = 0.0;
}


//  JR 2012-07-29
//  Perform beginning of b1pi calculation, just enough to determine those
//  amplitude which will be set to zero.  Amplitudes are set to (1,0) if 
//  they are not zero.  These amplitudes will need set to their correct
//  values on the call to GPUb1piAngAmp_kernel().
__global__ void Pretest_kernel( GPU_AMP_PROTO , int polBeam, GDouble polFrac,
  int J_X, int Par_X, int L_X, int I_X, int epsilon_R, int Iz_b1, int Iz_pi,
  GDouble u_rho_1, GDouble u_rho_3, GDouble u_omega_1, GDouble u_omega_3,
  GDouble u_b1_0, GDouble u_b1_2, 
  GDouble G0_omega, GDouble G0_b1, bool orthocheck) 
{
  // Calculate event for this thread.
  int iEvent = GPU_THIS_EVENT;
  WCUComplex CZero = { 0, 0 };
  WCUComplex COne  = { 1, 0 };

  int pol=(polBeam==1 ? +1 : -1); // y and x-pol. respectively
  
  //** (x)  This statement can be evaluated at top of function?
  if (J_X==0 && Par_X*pol*epsilon_R==-1) {
    pcDevAmp[iEvent] = CZero;
    return;
  }

  GDouble m0_omega = 0.783;
  GDouble m0_b1    = 1.223;
  bool isZero;


  //  Copy four-vectors for this thread from global memory.
  GDouble  b1s_pi   [4] = GPU_P4(3);
  GDouble  omegas_pi[4] = GPU_P4(4);
  GDouble  rhos_pim [4] = GPU_P4(5);
  GDouble  rhos_pip [4] = GPU_P4(6);

  //  Make four-vector sums
  GDouble  rho   [4] = ADD4(rhos_pip, rhos_pim );
  GDouble  omega [4] = ADD4(rho,     omegas_pi);
  GDouble  b1    [4] = ADD4(omega,   b1s_pi);

  //  Store mass of b1; for other vectors we can calculate mass on the fly.
  GDouble b1mass = MASS(b1);

  //  Is this term zero?
  isZero  = MASS(rho)+0.135            > m0_omega+3*G0_omega;
  isZero |= fabs(MASS(omega)-m0_omega) > 3*G0_omega;
  isZero |= fabs(b1mass-m0_b1)         > 3*G0_b1;
  isZero |= b1mass                     < (m0_omega - 3*G0_omega);
  if    (isZero) pcDevAmp[iEvent] = CZero;
  else           pcDevAmp[iEvent] = COne;
}






//  JR 2012-07-29
//  Calculate amplitudes only for those momenta sets with known non-zero
//  amplitudes.
__global__ void
GPUb1piAngAmp_kernel( 
  int cnt,
  // GPU_AMP_PROTO , 
  GDouble* pfDevData, WCUComplex* pcDevAmp, int* piDevPerm, int iNParticles, int iNEvents,
  int polBeam, GDouble polFrac,
  int J_X, int Par_X, int L_X, int I_X, int epsilon_R, int Iz_b1, int Iz_pi,
  GDouble u_rho_1, GDouble u_rho_3, GDouble u_omega_1, GDouble u_omega_3,
  GDouble u_b1_0, GDouble u_b1_2, 
  GDouble G0_omega, GDouble G0_b1, bool orthocheck) 
{

  // Calculate event for this thread.
  // int iEvent = GPU_THIS_EVENT;

  //  JR 2012-07-29
  //  NOTE:  This vesrsion of this function is called with different settings
  //         for threadIdx, blockIdx and blockDim than for the original version.
  //         The next line relects that change.
  int iEvent = threadIdx.x + blockIdx.x * blockDim.x;

  //  Skip this event index if it overruns number of events. 
  if (iEvent>=iNEvents) return;

  WCUComplex CZero = { 0, 0 };
  WCUComplex i =     { 0, 1 };
  WCUComplex COne =  { 1, 0 };

  int pol=(polBeam==1 ? +1 : -1); // y and x-pol. respectively
  
  if (J_X==0 && Par_X*pol*epsilon_R==-1) {
    pcDevAmp[iEvent] = CZero;
    return;
  }

  int m_X;
  GDouble u_rho, u_omega, u_b1;
  GDouble InvSqrt2 = 1.0/G_SQRT(2.0);
  GDouble m0_rho   = 0.775;
  GDouble G0_rho   = 0.149;
  GDouble m0_omega = 0.783;
  GDouble m0_b1    = 1.223;
  bool useCutoff   = true;
  bool isZero;

  //  Copy four-vectors for this thread from global memory.
  //  2012-05-19 JR  rhos_pip0,omega0,rho0 added for use
  //                 in BreitWigner_loc() below.
  GDouble  beam     [4] = GPU_P4(0);
  GDouble  recoil   [4] = GPU_P4(1);
  GDouble  Xs_pi    [4] = GPU_P4(2);
  GDouble  b1s_pi   [4] = GPU_P4(3);
  GDouble  omegas_pi[4] = GPU_P4(4);
  GDouble  rhos_pim [4] = GPU_P4(5);
  GDouble  rhos_pip [4] = GPU_P4(6);
  GDouble  rhos_pip0[4] = GPU_P4(6);

  //  Make four-vector sums
  GDouble  rho   [4] = ADD4(rhos_pip, rhos_pim );
  GDouble  rho0  [4] = ADD4(rhos_pip, rhos_pim );
  GDouble  omega [4] = ADD4(rho,     omegas_pi);
  GDouble  omega0[4] = ADD4(rho,     omegas_pi);
  GDouble  b1    [4] = ADD4(omega,   b1s_pi);


  //  Store mass of b1; for other vectors we can calculate mass on the fly.
  GDouble b1mass = MASS(b1);

  //  Is this term zero?
  if (useCutoff) {
      isZero  = MASS(rho)+0.135            > m0_omega+3*G0_omega;
      isZero |= fabs(MASS(omega)-m0_omega) > 3*G0_omega;
      isZero |= fabs(b1mass-m0_b1)         > 3*G0_b1;
      isZero |= b1mass                     < (m0_omega - 3*G0_omega);
      // Zero amplitude
      if (isZero) {
         pcDevAmp[iEvent] = CZero;
         return;
      }
  }

  // Continue to Calculate amplitude
  GDouble  X[4] = ADD4(b1,      Xs_pi);

  GDouble q = breakupMomentum( MASS(X), b1mass, MASS(Xs_pi) );

  GDouble alpha = phi( &(recoil[1]) );

  //  NOTE: Values of beam and recoil are changed below.
  boostToRest (beam,   X);
  boostToRest (recoil, X);

  //  Define new coordinate system with 
  //    - beam parallel to z direction
  //    - recoil in the x,z plain (i.e., y is normal to recoil and beam)
  //    - y is normal to beam and recoil.
  GDouble zGJ[3] = { beam[1], beam[2], beam[3] };
  makeUnit( zGJ );
  
  //**  (x)  Be care of cross order, need to check this
  // 2012-05-19 JR - Invert yGJ to make cross come out right.
  // GDouble yGJ[3] = { recoil[1], recoil[2], recoil[3] };
  GDouble yGJ[3] = { -recoil[1], -recoil[2], -recoil[3] };
  cross( yGJ, zGJ );
  makeUnit( yGJ );
  
  GDouble xGJ[3] = { yGJ[0], yGJ[1], yGJ[2] };
  cross( xGJ, zGJ );

  //particles to rest frames of their parents
  boostToRest (b1,      X);
  boostToRest (omega,   X);
  boostToRest (rho,     X);
  boostToRest (rhos_pip, X);
 
  // Note that in this form of the cascade of boosts, we are not
  // saving the 4-vecs in their intermediate RF, but going sequentially
  // straight to their immediate parent's RF.
  // Make sure to verify that the intermediares were not in fact needed
  // and that we didn't break anything with this simplification.
  MoveToRF(b1,omega);
  MoveToRF(b1,rho);      MoveToRF(omega,rho);
  MoveToRF(b1,rhos_pip); MoveToRF(omega,rhos_pip); MoveToRF(rho,rhos_pip);
  
  GDouble *b1_3vec=b1+1;
  GDouble ang_b1[]={dot(b1_3vec, xGJ),
                    dot(b1_3vec, yGJ),
                    dot(b1_3vec, zGJ)};
  GDouble b1_XRF_cosTheta = cosTheta(ang_b1);
  GDouble b1_XRF_phi      = phi(ang_b1);
   
  GDouble rho_omegaRF_cosTheta = cosTheta(rho+1);
  GDouble rho_omegaRF_phi      = phi(rho+1);
  GDouble rhos_pip_rhoRF_cosTheta = cosTheta(rhos_pip+1);
  GDouble rhos_pip_rhoRF_phi      = phi(rhos_pip+1);
  GDouble omega_b1RF_cosTheta     = cosTheta(omega+1);
  GDouble omega_b1RF_phi          = phi(omega+1);

/*
   List_l_R:        0 1 
   List_J_rho:        1 
   List_l_rho:   -1   1 
   List_L_omega:      1 
   List_l_omega: -1 0 1 
   List_L_b1:       0   2 
   List_l_b1:    -1 0 1 
*/

  // SUMMATION GUIDE:
  // notation meant to resemble TeX symbols in derivation
  // exception: pol = \epsilon_\gamma
  // l -> lambda, indicating helicity
  // u_[particle](q.n.) -> amplitude strength coefficient 

  int l_R_lim     = J_X + 1;
  
  //shortcut:  CB(L_X, J_b1, 0, l_b1 ; J_X, l_b1) vanishes when
  //  = CB(1, 1, 0, 0 ; 1, 0),  so omit l_b1=0 when J_X=L_X=1
  int l_b1_inc    = L_X==1 && J_X==1 ? 2 : 1;
  
  // restrict omega decay to just p wave
  int L_omega_lim = 1; // set to 3 to allow F wave
  int L_Rsign_lim;
  
  GDouble cosAlpha=G_COS(alpha), sinAlpha=G_SIN(alpha);
  WCUComplex expFact = {cosAlpha, sinAlpha};
  WCUComplex expFact_conj = {cosAlpha, -sinAlpha};
  
  WCUComplex ThelSum = { 0 , 0 };

  //  Setup dependent loop limits
  LOOP(l_gamma, -1, 1, 2) {

    
    LOOP(l_R, 0, l_R_lim, 1) {
      if(l_R==0 && epsilon_R==-1) continue;
    // LOOP(l_R, (1-epsilon_R)/2, l_R_lim, 1)  // if this still causes some GPU core
      // misalignment, try setting lower bound back to zero and tacking on
      //  * !(l_R==0 && epsilon_R==-1) 
      // to the long list of factors multiplying Thelsum below -IS


      //summing positive and negative helicity terms of R's reflectivity state
      L_Rsign_lim = l_R > 0 ? -1 : +1;
                        // Switch order of loop, because LOOP can only handle increasing increments
      // LOOP(l_Rsign, 1, L_Rsign_lim, -2) 
      LOOP(l_Rsign, L_Rsign_lim, 1, 2) {

        m_X = l_gamma - l_Rsign * l_R;
        if (m_X==0) {
          //testing for cancelation in |J 0>+pol*P*epsilon_R*(-1)^J|J 0>
          if(Par_X*pol*epsilon_R == (J_X % 2 ==0 ? -1:+1)) continue;
        } else {
          //enforcing that the selected projection <= vector magnitude 
          if( abs(m_X)>J_X) continue; 
        }
        
        
        WCUComplex l_b1DepTerm = {0,0};
        LOOP(l_b1, -1,1,l_b1_inc) {
          

          WCUComplex L_b1DepTerm = {0,0};

          LOOP(L_b1,0,2,2) {
            
          
            WCUComplex l_omegaDepTerm = {0,0};
            // 2012-05-19 JR  Fix l_omega loop
            // LOOP(l_omega,-1,0,1) 
            LOOP(l_omega,-1,1,1) {
              
              WCUComplex L_omegaDepTerm = {0,0};
              LOOP(L_omega, 1, L_omega_lim, 2) { 

                WCUComplex J_rhoDepTerm = {0,0};
                LOOP(J_rho, 1, L_omega_lim, 2) {

                  //enforces triang. ineq. betw. J_omega=1, J_rho and L_omega
                  // in effect, L_omega and J_rho take identical values
                  if( abs(J_rho-L_omega) > 1) continue; 
                
                
                  WCUComplex l_rhoDepTerm = {0,0};
                  LOOP(l_rho,-1,1,1) {
                    //shortcut CB(1,1,0,0;1,0)=0
                    if(L_omega==1 && J_rho==1 && l_rho==0) continue;
                  
                    l_rhoDepTerm += 
                      Conjugate(wignerD(1, l_omega, l_rho, 
                                        rho_omegaRF_cosTheta, rho_omegaRF_phi))
                      * clebsch(L_omega, 0, J_rho, l_rho, 1, l_rho)
                      * Y(J_rho, l_rho, rhos_pip_rhoRF_cosTheta, rhos_pip_rhoRF_phi);
                  }
                  
                  u_rho = J_rho==1 ? u_rho_1 : (J_rho==3 ? u_rho_3 : 0);
                  J_rhoDepTerm += u_rho * l_rhoDepTerm * 
                    BreitWigner_loc(m0_rho,G0_rho, J_rho,rhos_pip0,rhos_pim);
                }
                
                J_rhoDepTerm *= BreitWigner_loc(m0_omega, G0_omega, L_omega, omegas_pi,rho0);
                
                u_omega = L_omega==1 ? u_omega_1 : (L_omega==3 ? u_omega_3 : 0);
                L_omegaDepTerm += u_omega * J_rhoDepTerm * Nterm(L_omega);
              }
              
              l_omegaDepTerm += L_omegaDepTerm * 
                clebsch(L_b1, 0, 1, l_omega, 1, l_omega) *
                Conjugate(wignerD(1, l_b1, l_omega, 
                                  omega_b1RF_cosTheta, omega_b1RF_phi));
            }
            
            l_omegaDepTerm *= BreitWigner_loc(m0_b1, G0_b1, L_b1, b1s_pi, omega0);
            
            u_b1 = L_b1==0 ? u_b1_0 : (L_b1==2 ? u_b1_2 : 0); 
            L_b1DepTerm += u_b1 * l_omegaDepTerm * Nterm(L_b1);
          }
          //-- (_) understand why assignment here produces:
          // KERNEL LAUNCH ERROR [b1piAngAmp]: the launch timed out and was terminated
          // assigning/incrementing integers causes no problems
          
          l_b1DepTerm += L_b1DepTerm *
            Conjugate(wignerD(J_X, m_X, l_b1, b1_XRF_cosTheta, b1_XRF_phi)) *
            clebsch(L_X, 0, 1, l_b1, J_X, l_b1);
        }
        
        ThelSum += l_b1DepTerm  
          //to account for |eps_g> ~ (|1,-1>exp(-ia)-pol|1,+1>exp(ia)) 
          * (l_gamma==1 ? (-pol)*expFact : expFact_conj)
          //Assemble reflectivity eigenvector with epsilon_X=pol*epslion_R
          * (GDouble) (m_X<0 ? Par_X*pol*epsilon_R*((J_X-m_X) % 2 == 0 ? +1:-1) : 1) 
          * (GDouble) (m_X == 0 ? 1.0 : InvSqrt2 )
          // to apply th(l_R) reflectivity state prefactor: 
          // m=0: 1/2  m>0: 1/sqrt(2)  m<0: 0 (last just skipped in this sum)  
          * (GDouble) (l_R > 0 ? InvSqrt2 : 1.0 )
          //apply coefficients to the reflectivity basis terms:
          * (GDouble) (l_Rsign==1 ? 1 : epsilon_R)
          ; //v(*epsilon_R) *     
        
      }
    }
  }
  
  ThelSum *= Nterm(L_X) * 
    // barrier factor
//    (GDouble)(L_X==0 ? 1.0 : (L_X==1 ? q : G_POW(q,L_X))) *
    (GDouble)(L_X==0 ? 1.0 : (L_X==1 ? q : ::pow(q,L_X))) *
    // to apply polarization fraction weights: 
    (GDouble)G_SQRT((1.0-pol*polFrac)*0.5) * //(1+g) for x-pol, (1-g) for y-pol   
    (pol==1 ? i : COne)*InvSqrt2 * //to account for |eps_g> ~ sqrt(-eps/2)
    clebsch(1, Iz_b1, 1, Iz_pi, I_X, Iz_b1 + Iz_pi);
  pcDevAmp[iEvent] = ThelSum;

  
}





#ifdef DEBUG
//   This is for debugging 
//     It reads the amplitdues and momemta vectors from the CUDA device and prints them.
void 
printCudaArrays(GDouble* pfDevData, WCUComplex* pcDevAmp, int* piDevPerm, int iNParticles, int iNEvents, int cnt) {

  //  Read amplitudes from GPU to CPU
	GDouble *amp = (GDouble *) malloc (iNEvents * 2 * sizeof(GDouble));
	hipMemcpy (amp, pcDevAmp, iNEvents * 2 * sizeof(GDouble), hipMemcpyDeviceToHost);

  //  Copy 4momenta from GPU to CPU - make part() big enough to hold the entire set of momenta
  GDouble *part = (GDouble *) malloc (iNEvents * 4 * iNParticles * sizeof(GDouble));
  hipMemcpy (part, pfDevData, iNEvents * 4 * iNParticles * sizeof(GDouble), hipMemcpyDeviceToHost);

	//  Print arrays
	int ievent, ipart, idim;
	int ndim = 4;
	for (ievent=0; ievent<iNEvents; ievent++) {
		printf ("test: CUDA: %2d %6d ", cnt, ievent);
		//  Print amplitude
		printf ("  %12.4e %12.4e", amp[2*ievent], amp[2*ievent+1]);
		for (ipart=0;ipart<iNParticles;ipart++) {
			printf (" ");
			for (idim=0;idim<4;idim++) {
				printf ( " %8.4f", part[ievent + idim*iNEvents + ipart*ndim*iNEvents ] );
			}
		}
		printf("\n");
	}

	//  Free allocations from arrays allocated withing this function
	if (amp)  free(amp);
	if (part) free(part);

}
#endif


void
GPUb1piAngAmp_exec(dim3 dimGrid, dim3 dimBlock, 
									 // GPU_AMP_PROTO,
                   GDouble* pfDevData, WCUComplex* pcDevAmp, int* piDevPerm, int iNParticles, int iNEvents,
                   int polBeam, GDouble polFrac,
                   int J_X, int Par_X, int L_X, int I_X, int epsilon_R, 
                   int Iz_b1, int Iz_pi,
                   GDouble u_rho_1, GDouble u_rho_3, 
                   GDouble u_omega_1, GDouble u_omega_3,
                   GDouble u_b1_0, GDouble u_b1_2, 
                   GDouble G0_omega, GDouble G0_b1, bool orthocheck)
{  
  int ievent, ievent1, idim, ipart, i, j, k;
  int nonZero = 0;
  int static cnt = 0;
	cnt++;
// printf("test: Call to GPUb1piAngAmp_exec: cnt %d\n", cnt);

  // Identify amplitudes which are zero
  Pretest_kernel<<< dimGrid, dimBlock >>>
    ( 
      // GPU_AMP_ARGS, 
      pfDevData, pcDevAmp, piDevPerm, iNParticles, iNEvents,
      polBeam, polFrac, 
      J_X, Par_X, L_X, I_X, epsilon_R, Iz_b1, Iz_pi,
      u_rho_1, u_rho_3, u_omega_1, u_omega_3, u_b1_0, u_b1_2, 
      G0_omega, G0_b1, orthocheck ); 
// printf("test: after call to Pretest_kernel()\n");


  //  Copy pcDevAmp from device to host  */
  GDouble *hostAmp = (GDouble *) malloc(2*iNEvents*sizeof(GDouble));
  hipMemcpy (hostAmp, pcDevAmp, 2*iNEvents*sizeof(GDouble), hipMemcpyDeviceToHost);

  //  Initialize all on-device amplitudes to zero
  Setzero_kernel<<< dimGrid, dimBlock >>>(pcDevAmp,iNEvents);
// printf("test: after call to Setzero_kernel()\n");
  

  //  Count number of nonZero amplitudes
  for (i=0;i<iNEvents;i++) {
    if (hostAmp[2*i]==1.0) nonZero++;
  }

  //  Allocate array to hold indices of nonZero amplitudes
  int *nonZeroIndices = (int *) malloc(nonZero * sizeof(int));
  j = 0;
  for (i=0;i<iNEvents;i++) {
    if (hostAmp[2*i]==1.0) nonZeroIndices[j++] = i;
  }

  //  Copy 4momenta from GPU to CPU - make part() big enough to hold the entire set of momenta
  GDouble *part = (GDouble *) malloc (iNEvents * 4 * iNParticles * sizeof(GDouble));
  hipMemcpy (part, pfDevData, iNEvents * 4 * iNParticles * sizeof(GDouble), hipMemcpyDeviceToHost);
// printf("test: after copy pfDevData to Device\n");

  //  Copy nonZero momenta in place to the start of the array part
  //   Make sure order of copying moves continuously from lower to higher indice.
  for (ipart=0;ipart<iNParticles;ipart++) {
    for (idim=0;idim<4;idim++) {
      for (ievent1=0;ievent1<nonZero;ievent1++) {
        ievent = nonZeroIndices[ievent1];
        //  Index of nonZero event in original particle array
        i = ievent  + idim * iNEvents + ipart * 4 * iNEvents;
        //  Index of nonZero event in new      particle array
        j = ievent1 + idim * nonZero  + ipart * 4 * nonZero;
        part[j] = part[i];
      }
    }
  }



  //  Copy new particles on CPU back to GPU, only need those momenta sets which were non-zero, not the size of the entire set.
	GDouble *part_dev;
  hipMalloc(&part_dev,       nonZero * 4 * iNParticles * sizeof(GDouble) );
  hipMemcpy( part_dev, part, nonZero * 4 * iNParticles * sizeof(GDouble), hipMemcpyHostToDevice );
// printf("test: after copy Part to Device\n");
  

  //  Reset dimGrid and dimBlock for the value of nonZero
  int Nthreads = 32;
  dim3 dimBlock1(Nthreads);
  dim3 dimGrid1((nonZero-1)/Nthreads+1);
  

  //   Evaluate non-zero amplitudes
  // iNEvents = nonZero;
  GPUb1piAngAmp_kernel<<< dimGrid1, dimBlock1 >>>
    ( 
      cnt, 
      // GPU_AMP_ARGS, 
      // pfDevData, pcDevAmp, piDevPerm, iNParticles, nonZero,
      part_dev, pcDevAmp, piDevPerm, iNParticles, nonZero,
      polBeam, polFrac, 
      J_X, Par_X, L_X, I_X, epsilon_R, Iz_b1, Iz_pi,
      u_rho_1, u_rho_3, u_omega_1, u_omega_3, u_b1_0, u_b1_2, 
      G0_omega, G0_b1, orthocheck ); 
// printf("test: after call to GUPb1piAngAmp_kernel()\n");

  //  Read amplitudes from GPU to CPU
	GDouble *amp = (GDouble *) malloc (iNEvents * 2 * sizeof(GDouble));
	hipMemcpy (amp, pcDevAmp, iNEvents * 2 * sizeof(GDouble), hipMemcpyDeviceToHost);

// printf("test: after copy Amp to Host\n");


  //  Re-arrange location of amplitudes on GPU to match original distribution of vectors
	//  Progress through the index array backward.
	k = iNEvents;
	for (i=nonZero-1;i>=0;i--) {
		//  Zero those elements between this element and last.
		for (j=nonZeroIndices[i]+1;j<k;j++) {
			amp[2*j  ] = 0.0;
			amp[2*j+1] = 0.0;
		}
		k = nonZeroIndices[i];
		amp[2*k  ] = amp[2*i  ];
		amp[2*k+1] = amp[2*i+1];
	}
	//  Zero remaining elements
	for (j=0;j<nonZeroIndices[0];j++) {
			amp[2*j  ] = 0.0;
			amp[2*j+1] = 0.0;
	}

	//  Write values back to GPU so calling program will find them where they
	//  expect them.
	hipMemcpy (pcDevAmp, amp, iNEvents * 2 * sizeof(GDouble), hipMemcpyHostToDevice);
// printf("test: after copy Amp to Device\n");
	
	//  Free allocations
  if (part_dev) hipFree(part_dev);
	if (amp)  free(amp);
	if (part) free(part);

// printf("test: after Free allocations\n");
//  Print Particle and Amplitude CUDA arrays
#ifdef DEBUG
printCudaArrays(pfDevData, pcDevAmp, piDevPerm, iNParticles, iNEvents, cnt);
#endif

}
